#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include "hipblas.h"
#include<cassert>

int main(int argc, char* argv[]) {
    hipblasHandle_t handle;
    const int n = 1e8;
    const double alpha = 1.0;
    double* h_x = new double[n];
    double* h_y = new double[n];
    double* h_z = new double[n];
    double *x, *y, *z;

    hipMalloc(&x, n*sizeof(double));
    hipMalloc(&y, n*sizeof(double));
    hipMalloc(&z, n*sizeof(double));

    hipblasCreate(&handle);

    // Initialize x and y on the host
    for(int i=0; i<n; i++) {
        h_x[i] = i;
        h_y[i] = n-i;
    }

    // Copy x and y to device
    hipMemcpy(x, h_x, n*sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(y, h_y, n*sizeof(double), hipMemcpyHostToDevice);

    // Compute sum of x and y
    hipblasDcopy(handle, n, x, 1, z, 1);
    hipblasDaxpy(handle, n, &alpha, y, 1, z, 1);

    // Copy z to host
    hipMemcpy(h_z, z, n*sizeof(double), hipMemcpyDeviceToHost);

    // Assert that the sum is correct
    for(int i=0; i<n; i++) {
        assert(h_z[i] == n);
    }

    delete[] h_x, h_y, h_z;
    hipFree(x);
    hipFree(y);
    hipFree(z);

    hipblasDestroy(handle);

    return 0;
}